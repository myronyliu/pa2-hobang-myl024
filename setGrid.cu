
#include <hip/hip_runtime.h>
void setGrid(int n, dim3 &blockDim, dim3 &gridDim)
{
	// set your block dimensions and grid dimensions here
	gridDim.x = n / 16;
	gridDim.y = n / 64;

	if(n % blockDim.x != 0)
	{
		gridDim.x++;
	}

	if(n % blockDim.y != 0)
	{
		gridDim.y++;
	}
}
