#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include "utils.h"
#include "types.h"
using namespace std;

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int wA = N;
    int wB = N;
    const int BLOCK_SIZE = 32; //square

    int aBegin = wA * BLOCK_SIZE * by;
    int aEnd   = aBegin + wA - 1;
    int aStep  = BLOCK_SIZE;
    int bBegin = BLOCK_SIZE * bx;
    int bStep  = BLOCK_SIZE * wB;

    _DOUBLE_ Csub = 0;

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
    {
		__shared__ _DOUBLE_ As[BLOCK_SIZE][BLOCK_SIZE];
    	__shared__ _DOUBLE_ Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        __syncthreads();

		#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}
